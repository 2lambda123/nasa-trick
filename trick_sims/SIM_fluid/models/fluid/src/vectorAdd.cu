#include <stdio.h>
#include "hip/hip_runtime.h"
#define SIZE 1024


__device__ int add(int a, int b) {
	return a + b;
}
__global__ void VectorAdd(int *a, int *b, int *c, int n) 
{
	int i = threadIdx.x;
	if (i < n)
		c[i] = add(a[i], b[i]);
}

void callVectorAdd() 
{
	int *a, *b, *c;
	
	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));
	
	for(int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}
	VectorAdd <<<1, SIZE>>>(a, b, c, SIZE);
	hipDeviceSynchronize();
	for (int i = 0; i < 10; ++i)
		printf("c[%d] = %d\n", i, c[i]);
		
	hipFree(a);
	hipFree(b);
	hipFree(c);

}