#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include "sph_gpu.h"


#define NUM_THREADS 1024

bool particlesOnGPU = false;
Particle* d_particles;
Fluid* d_fluid;
int* d_n;


__global__ void computeDensityAndPressureGPU(Particle* particles, int* n, Fluid* fluid) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;

	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		pi.rho = 0;
		//Particle* candidate_neighbors = all_neighbors[i];
		for (int j = 0; j < *n; j++) {
			//Particle & pj = candidate_neighbors[j];
			Particle& pj = particles[j];
			float rij[3] = {pj.pos[0] - pi.pos[0], pj.pos[1] - pi.pos[1], pj.pos[2] - pi.pos[2]};
			float r = sqrt(rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);
			if (r >= 0 && r <= fluid->H) {
				pi.rho += fluid->MASS * fluid->POLY6 * pow(fluid->HSQ - r * r, 3.f);
			}
		}
		pi.pressure = fluid->GAS_CONST * (pi.rho - fluid->REST_DENS);
	}
}



__global__ void computeForcesGPU(Particle* particles, int* n, Fluid* fluid) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;

	//for(auto &pi : particles) {
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		float pressure_force[3] = {0, 0, 0};
		float viscosity_force[3] = {0, 0, 0};
		//Particle* candidate_neighbors = all_neighbors[i];
		for (int j = 0; j < *n; j++) {
			//Particle& pj = candidate_neighbors[j];
			Particle& pj = particles[j];
			if (&pi != &pj) {
				float rij[3] = {pj.pos[0] - pi.pos[0], pj.pos[1] - pi.pos[1], pj.pos[2] - pi.pos[2]};
				float r = std::sqrt(rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);
				float rij_hat[3] = {rij[0] / r, rij[1] / r, rij[2] / r};
				if (r > 0 && r <= fluid->H) {
					pressure_force[0] -= rij_hat[0] * fluid->MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * fluid->SPIKY_GRAD * pow(fluid->H - r, 2.f);
					pressure_force[1] -= rij_hat[1] * fluid->MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * fluid->SPIKY_GRAD * pow(fluid->H - r, 2.f);
					pressure_force[2] -= rij_hat[2] * fluid->MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * fluid->SPIKY_GRAD * pow(fluid->H - r, 2.f);

					viscosity_force[0] += fluid->VISC * fluid->MASS * ((pj.velocity[0] - pi.velocity[0]) / pj.rho) * fluid->VISC_LAP * (fluid->H - r);
					viscosity_force[1] += fluid->VISC * fluid->MASS * ((pj.velocity[1] - pi.velocity[1]) / pj.rho) * fluid->VISC_LAP * (fluid->H - r);
					viscosity_force[2] += fluid->VISC * fluid->MASS * ((pj.velocity[2] - pi.velocity[2]) / pj.rho) * fluid->VISC_LAP * (fluid->H - r);


				}
			}
		}
		float G[3] = {0.f, fluid->G_STRENGTH * -9.8f, 0.f};
		float gravity_force[3] = {pi.rho * G[0], pi.rho * G[1], pi.rho * G[2]};
		pi.force[0] = viscosity_force[0] + pressure_force[0] + gravity_force[0];
		pi.force[1] = viscosity_force[1] + pressure_force[1] + gravity_force[1];
		pi.force[2] = viscosity_force[2] + pressure_force[2] + gravity_force[2];
	}

}
/*
__global__ void verletUpdatePosition(Particle* particles, int* n) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		pi.pos += DT * pi.velocity;
	}
}
*/
__global__ void timeIntegrationGPU(Particle* particles, int* n, Fluid* fluid) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		//pi.pos += DT * pi.velocity;
		pi.velocity[0] += fluid->DT * pi.force[0] / pi.rho;
		pi.velocity[1] += fluid->DT * pi.force[1] / pi.rho;
		pi.velocity[2] += fluid->DT * pi.force[2] / pi.rho;

		if (pi.pos[2] - fluid->EPS < -fluid->BOUND) {
			pi.velocity[2] *= fluid->BOUND_DAMPING;
			pi.pos[2] = -fluid->BOUND + fluid->EPS;
		}

		if (pi.pos[2] + fluid->EPS > fluid->BOUND) {
			pi.velocity[2] *= fluid->BOUND_DAMPING;
			pi.pos[2] = fluid->BOUND - fluid->EPS;
		}

		if (pi.pos[1] - fluid->EPS < -fluid->BOUND) {
			pi.velocity[1] *= fluid->BOUND_DAMPING;
			pi.pos[1] = -fluid->BOUND + fluid->EPS;
		}

		if (pi.pos[1] + fluid->EPS > fluid->BOUND) {
			pi.velocity[1] *= fluid->BOUND_DAMPING;
			pi.pos[1] = fluid->BOUND - fluid->EPS;
		}

		if (pi.pos[0] - fluid->EPS < -fluid->BOUND) {
			pi.velocity[0] *= fluid->BOUND_DAMPING;
			pi.pos[0] = -fluid->BOUND + fluid->EPS;
		}
		if (pi.pos[0] + fluid->EPS > fluid->BOUND) {
			pi.velocity[0] *= fluid->BOUND_DAMPING;
			pi.pos[0] = fluid->BOUND - fluid->EPS;
		}
	}

}

void updateSPH_GPU(std::vector<Particle>& particles, Fluid* fluid) {
	int n = fluid->NUM_PARTICLES;
	
	if (!particlesOnGPU) {
		hipMalloc(&d_particles, n * sizeof(Particle));
		hipMalloc(&d_fluid, sizeof(Fluid));
		hipMalloc(&d_n, sizeof(int));

		hipMemcpy(d_fluid, fluid, sizeof(Fluid), hipMemcpyHostToDevice);
		hipMemcpy(d_particles, particles.data(), n * sizeof(Particle), hipMemcpyHostToDevice);
		hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
		particlesOnGPU = true;
	}

	//verletUpdatePosition<<<1, NUM_THREADS>>>(d_particles, d_n);

	computeDensityAndPressureGPU << <1, NUM_THREADS >> > (d_particles, d_n, d_fluid);
	
	computeForcesGPU << <1, NUM_THREADS >> > (d_particles, d_n, d_fluid);
	
	timeIntegrationGPU << <1, NUM_THREADS >> > (d_particles, d_n, d_fluid);

	hipDeviceSynchronize();

	hipMemcpy(particles.data(), d_particles, n * sizeof(Particle), hipMemcpyDeviceToHost);


}

