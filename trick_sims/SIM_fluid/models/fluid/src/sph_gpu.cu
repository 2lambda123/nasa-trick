#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include "sph_gpu.h"


#define NUM_THREADS 1024

bool particlesOnGPU = false;
Particle* d_particles;
Fluid* d_fluid;
int* d_n;


__global__ void computeDensityAndPressureGPU(Particle* particles, int* n, Fluid* fluid) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;

	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		pi.rho = 0;
		//Particle* candidate_neighbors = all_neighbors[i];
		for (int j = 0; j < *n; j++) {
			//Particle & pj = candidate_neighbors[j];
			Particle& pj = particles[j];
			float rij[3] = {pj.pos[0] - pi.pos[0], pj.pos[1] - pi.pos[1], pj.pos[2] - pi.pos[2]};
			float r = sqrt(rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);
			if (r >= 0 && r <= fluid->H) {
				pi.rho += fluid->MASS * fluid->POLY6 * pow(fluid->HSQ - r * r, 3.f);
			}
		}
		pi.pressure = fluid->GAS_CONST * (pi.rho - fluid->REST_DENS);
	}
}

/*

__global__ void computeForcesGPU(Particle* particles, int* n) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;

	//for(auto &pi : particles) {
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		glm::vec3 pressure_force(0, 0, 0);
		glm::vec3 viscosity_force(0, 0, 0);
		//Particle* candidate_neighbors = all_neighbors[i];
		for (int j = 0; j < *n; j++) {
			//Particle& pj = candidate_neighbors[j];
			Particle& pj = particles[j];
			if (&pi != &pj) {
				glm::vec3 rij = pj.pos - pi.pos;
				float r = glm::length(rij);
				glm::vec3 rij_hat = glm::normalize(rij);
				if (r > 0 && r <= H) {
					pressure_force -= rij_hat * MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * SPIKY_GRAD * pow(H - r, 2.f);

					viscosity_force += VISC * MASS * ((pj.velocity - pi.velocity) / pj.rho) * VISC_LAP * (H - r);


				}
			}
		}
		glm::vec3 G(0.f, G_STRENGTH * -9.8f, 0.f);
		glm::vec3 gravity_force = pi.rho * G;
		pi.force = viscosity_force + pressure_force + gravity_force;
	}

}
__global__ void verletUpdatePosition(Particle* particles, int* n) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		pi.pos += DT * pi.velocity;
	}
}

__global__ void timeIntegrationGPU(Particle* particles, int* n) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		//pi.pos += DT * pi.velocity;
		pi.velocity += DT * pi.force / pi.rho;

		if (pi.pos.z - EPS < -BOUND) {
			pi.velocity.z *= BOUND_DAMPING;
			pi.pos.z = -BOUND + EPS;
		}

		if (pi.pos.z + EPS > BOUND) {
			pi.velocity.z *= BOUND_DAMPING;
			pi.pos.z = BOUND - EPS;
		}

		if (pi.pos.y - EPS < -BOUND) {
			pi.velocity.y *= BOUND_DAMPING;
			pi.pos.y = -BOUND + EPS;
		}

		if (pi.pos.y + EPS > BOUND) {
			pi.velocity.y *= BOUND_DAMPING;
			pi.pos.y = BOUND - EPS;
		}

		if (pi.pos.x - EPS < -BOUND) {
			pi.velocity.x *= BOUND_DAMPING;
			pi.pos.x = -BOUND + EPS;
		}
		if (pi.pos.x + EPS > BOUND) {
			pi.velocity.x *= BOUND_DAMPING;
			pi.pos.x = BOUND - EPS;
		}
	}

}*/

void updateSPH_GPU(std::vector<Particle>& particles, Fluid* fluid) {
	int n = fluid->NUM_PARTICLES;
	
	if (!particlesOnGPU) {

		

		hipMalloc(&d_particles, n * sizeof(Particle));
		hipMalloc(&d_fluid, sizeof(Fluid));
		hipMalloc(&d_n, sizeof(int));

		hipMemcpy(d_fluid, fluid, sizeof(Fluid), hipMemcpyHostToDevice);
		hipMemcpy(d_particles, particles.data(), n * sizeof(Particle), hipMemcpyHostToDevice);
		hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
		particlesOnGPU = true;
	}

	//verletUpdatePosition<<<1, NUM_THREADS>>>(d_particles, d_n);

	computeDensityAndPressureGPU << <1, NUM_THREADS >> > (d_particles, d_n, d_fluid);
	
	//computeForcesGPU << <1, NUM_THREADS >> > (d_particles, d_n);
	
	//timeIntegrationGPU << <1, NUM_THREADS >> > (d_particles, d_n);

	hipDeviceSynchronize();

	hipMemcpy(particles.data(), d_particles, n * sizeof(Particle), hipMemcpyDeviceToHost);


}

