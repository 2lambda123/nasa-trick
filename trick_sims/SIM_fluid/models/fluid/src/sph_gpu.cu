#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "sph_gpu.cuh"


#define NUM_THREADS 1024
bool particlesOnGPU = false;
Particle* d_particles;
int* d_n;


__global__ void computeDensityAndPressureGPU(Particle* particles, int* n) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;

	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		pi.rho = 0;
		//Particle* candidate_neighbors = all_neighbors[i];
		for (int j = 0; j < *n; j++) {
			//Particle & pj = candidate_neighbors[j];
			Particle& pj = particles[j];
			glm::vec3 rij = pj.pos - pi.pos;
			float r = glm::length(rij);
			if (r >= 0 && r <= H) {
				pi.rho += MASS * POLY6 * pow(HSQ - r * r, 3.f);
			}
		}
		pi.pressure = GAS_CONST * (pi.rho - REST_DENS);
	}
}

__global__ void computeForcesGPU(Particle* particles, int* n) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;

	//for(auto &pi : particles) {
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		glm::vec3 pressure_force(0, 0, 0);
		glm::vec3 viscosity_force(0, 0, 0);
		//Particle* candidate_neighbors = all_neighbors[i];
		for (int j = 0; j < *n; j++) {
			//Particle& pj = candidate_neighbors[j];
			Particle& pj = particles[j];
			if (&pi != &pj) {
				glm::vec3 rij = pj.pos - pi.pos;
				float r = glm::length(rij);
				glm::vec3 rij_hat = glm::normalize(rij);
				if (r > 0 && r <= H) {
					pressure_force -= rij_hat * MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * SPIKY_GRAD * pow(H - r, 2.f);

					viscosity_force += VISC * MASS * ((pj.velocity - pi.velocity) / pj.rho) * VISC_LAP * (H - r);


				}
			}
		}
		glm::vec3 G(0.f, G_STRENGTH * -9.8f, 0.f);
		glm::vec3 gravity_force = pi.rho * G;
		pi.force = viscosity_force + pressure_force + gravity_force;
	}

}
__global__ void verletUpdatePosition(Particle* particles, int* n) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		pi.pos += DT * pi.velocity;
	}
}

__global__ void timeIntegrationGPU(Particle* particles, int* n) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		//pi.pos += DT * pi.velocity;
		pi.velocity += DT * pi.force / pi.rho;

		if (pi.pos.z - EPS < -BOUND) {
			pi.velocity.z *= BOUND_DAMPING;
			pi.pos.z = -BOUND + EPS;
		}

		if (pi.pos.z + EPS > BOUND) {
			pi.velocity.z *= BOUND_DAMPING;
			pi.pos.z = BOUND - EPS;
		}

		if (pi.pos.y - EPS < -BOUND) {
			pi.velocity.y *= BOUND_DAMPING;
			pi.pos.y = -BOUND + EPS;
		}

		if (pi.pos.y + EPS > BOUND) {
			pi.velocity.y *= BOUND_DAMPING;
			pi.pos.y = BOUND - EPS;
		}

		if (pi.pos.x - EPS < -BOUND) {
			pi.velocity.x *= BOUND_DAMPING;
			pi.pos.x = -BOUND + EPS;
		}
		if (pi.pos.x + EPS > BOUND) {
			pi.velocity.x *= BOUND_DAMPING;
			pi.pos.x = BOUND - EPS;
		}
	}

}

void updateSPH_GPU(std::vector<Particle> &particles) {

	if (!particlesOnGPU) {

		int n = NUM_PARTICLES;

		hipMalloc(&d_particles, NUM_PARTICLES * sizeof(Particle));

		hipMalloc(&d_n, sizeof(int));

		hipMemcpy(d_particles, particles.data(), NUM_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);
		hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
		particlesOnGPU = true;
	}

	verletUpdatePosition<<<1, NUM_THREADS>>>(d_particles, d_n);

	computeDensityAndPressureGPU << <1, NUM_THREADS >> > (d_particles, d_n);

	computeForcesGPU << <1, NUM_THREADS >> > (d_particles, d_n);
	
	timeIntegrationGPU << <1, NUM_THREADS >> > (d_particles, d_n);

	hipDeviceSynchronize();

	hipMemcpy(particles.data(), d_particles, NUM_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost);


}

